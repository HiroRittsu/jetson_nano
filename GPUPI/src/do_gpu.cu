#include "hip/hip_runtime.h"
//====================================================
// Calculate any digit of Pi using BBP formula
//     pi_gpu.cu : Calculate Pi by GPU (Program)
//----------------------------------------------------
// Rev.01 2019.04.28 M.Munetomo
//----------------------------------------------------
// Copyright (C) 2019 Munetomo Maruyama
//====================================================

#include <cinttypes>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <sys/time.h>

#include "common.h"
#include "do_gpu.h"

//--------------------------------
// Constant Hex Table
//--------------------------------
__constant__ char CM_hex_table[16] = { '0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 'A', 'B', 'C', 'D', 'E', 'F' };

//----------------------------------------------
// Binary Modulo Exponentiation r=(a^b)mod(c)
//----------------------------------------------
__device__ int64_t GPU_Bin_Mod_Exp(int64_t a, int64_t b, int64_t c)
{
    int64_t ret = 1;
    while (b != 0)
    {
        if (b % 2)
        {
            ret = (ret * a) % c;
        }
        a = (a * a) % c;
        b = b / 2;
    }
    return ret;
}

//------------------------------
// Calculate S(j, d)
//------------------------------
__device__ double GPU_Sjd(int64_t j, int64_t d)
{
    double sum = 0;
    // k = 0 ... d
    for (int64_t k = 0; k <= d; k++)
    {
        sum = sum + (double)GPU_Bin_Mod_Exp(16, d - k, 8 * k + j) / (double)(8 * k + j);
    }
    // k = (d + 1) ...
    double numerator = 1;
    double denominator = 8 * d + j;
    double increase;
    for (int64_t k = 0; k < 8; k++)
    {
        numerator = numerator / 16;
        denominator = denominator + 8;
        increase = numerator / denominator;
        sum = sum + increase;
    }
    //    
    sum = sum - (int)sum;  // extract decimal part
    //
    return sum;
}

//-----------------------------
// Whole Number to Hex
//-----------------------------
__device__ char GPU_WholeNum_to_Hex(double input)
{
    int index = (int) input;
    if ((index < 0) || (index > 15)) return '*';
    return CM_hex_table[index];
}

//------------------------------------------
// Calculate Pi at specified digit
//------------------------------------------
__global__ void GPU_Calc_Pi_kernel(uint64_t digit_max, char *result_hex)
{
    // Which digit should I calculate?
    uint64_t ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (ix >= digit_max / DIGIT_STEP) return;
    int64_t digit = ix * DIGIT_STEP;
    //
    double Pi16d = 0;
    //
    Pi16d = 4 * GPU_Sjd(1, digit)
          - 2 * GPU_Sjd(4, digit)
          - 1 * GPU_Sjd(5, digit)
          - 1 * GPU_Sjd(6, digit);
    //
    Pi16d = (Pi16d > 0) ? (Pi16d - (int) Pi16d) : (Pi16d - (int) Pi16d + 1);
    //
    for (int i = 0; i < DIGIT_STEP; i++)
    {
        Pi16d = Pi16d * 16;
        *(result_hex + digit + i) = GPU_WholeNum_to_Hex(Pi16d);
        Pi16d = Pi16d - (int) Pi16d;
    }
}

//--------------------------------
// GPU Main Routine
//--------------------------------
void GPU_Main(int64_t digit)
{
    //
    // Start Message
    printf("--------[GPU] %lld digits Calculation...(showing specified last %d digits)\n", (long long int)digit, DIGIT_SHOW);
    //
    // Adjust Digits
    int64_t digit_show_bgn = (digit < DIGIT_SHOW)? 0 : digit -DIGIT_SHOW; // digit to be shown
    digit = ((digit + DIGIT_STEP - 1) / DIGIT_STEP) * DIGIT_STEP; // digit to be calculated
    //
    // Allocate Result Buffer
    char *h_gpu_result = NULL;
    h_gpu_result = (char*) malloc((digit + DIGIT_STEP) * sizeof(char));
    if (h_gpu_result == NULL) exit(EXIT_FAILURE);
    memset(h_gpu_result, 0, digit + DIGIT_STEP);
    //
    char *d_gpu_result = NULL;
    CHECK(hipMalloc((void **) &d_gpu_result, (digit + DIGIT_STEP) * sizeof(char)));
    CHECK(hipMemcpy(d_gpu_result, h_gpu_result, digit, hipMemcpyHostToDevice));        
    //
    // Calculate Pi by BBP formula
    double iStart = CPU_Second();
    dim3 block(256);
    dim3 grid((digit / DIGIT_STEP + block.x - 1) / block.x);
    GPU_Calc_Pi_kernel<<<grid, block>>>(digit, d_gpu_result);
    CHECK(hipDeviceSynchronize());
    double iElaps = CPU_Second() - iStart;
    //
    // Display Result
    CHECK(hipMemcpy(h_gpu_result, d_gpu_result, digit, hipMemcpyDeviceToHost));
    int64_t d;
    for (d = digit_show_bgn; d < digit_show_bgn + DIGIT_SHOW; d++)
    {
        printf("%c", h_gpu_result[d]);
    }
    printf("\n");
    printf("Time elapsed %lf sec\n", iElaps);    
    //
    // Finish
    CHECK(hipFree(d_gpu_result));
    CHECK(hipDeviceReset());
    free(h_gpu_result);
}

//====================================================
// End of Program
//====================================================
